#pragma once
#include <hip/hip_runtime.h>

#include<iostream>
#include<math.h>


__global__ void matMulKernel(
    float *a, int a_rows, int a_cols, 
    float *b, int b_rows, int b_cols,
    float *c, int c_rows, int c_cols
)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	
    for (int i = 0; i < a_cols; ++i)
	{
		Cvalue += a[row * a_cols + i] * b[i * b_cols + col];
	}

    c[row * c_cols + col] = Cvalue;
}

/*
    Here is the first version
    
    array A is the input matrix
    array B is the projection matrix
    array C is the results matrix

    __global__ void hash_(
        float *A, int a_rows, int a_cols, 
        float *B, int b_rows, int b_cols,
        float *C, int c_rows, int c_cols, float* bits
    )
    {
        float Cvalue = 0.0;
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        
        int table_index = row / a_rows;
        int input_index = row % a_rows;

        for (int i = 0; i < a_cols; ++i)
        {
            Cvalue += A[input_index * a_cols + i] * B[table_index * (b_rows * b_cols) + i * b_cols + col];
        }

        C[row * c_cols + col] = std::signbit(-1 * Cvalue) * bits[col];
        // C[row * c_cols + col] = Cvalue;
    }

    __global__ void vec_sum(float* a, float* b, int cols) {
        int row = threadIdx.x + blockIdx.x * blockDim.x;

        for (int j = 0; j < cols; ++j) {
            b[row] += a[row * cols + j];
        }
        
    }
*/


__global__ void hash_(
    float *A, int a_rows, int a_cols, 
    float *B, int b_rows, int b_cols,
    float *C, int c_rows, int c_cols, float* bits
)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (row < a_rows && col < c_cols){
        int table_index = col / b_cols;
        int input_index = row;

        for (int i = 0; i < a_cols; ++i)
        {
            Cvalue += A[input_index * a_cols + i] * B[table_index * (b_rows * b_cols) + i * b_cols + col % b_cols];
        }
        C[row * c_cols + col] = std::signbit(-1 * Cvalue) * bits[col % b_cols];
    }
}


__global__ void vec_sum(float* a, float* b, int input_size, int table_nums, int hash_size) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;    
    
    int table_index = blockIdx.x;

    if (row < input_size && table_index < table_nums) {

        for (int j = 0; j < hash_size; ++j) {
            b[row + table_index * input_size] += a[row * (table_nums * hash_size) + j + table_index * hash_size];
        }
    }
}