#include<hip/hip_runtime.h>
#include "kernel.cu"
#include<iostream>

#include<chrono> 
#include<vector>
#include<string.h>
#include "helper.cu"

using namespace std;
using namespace std::chrono; 


void index(float* input, int inputSize, int inputDim, // scale of input
    float* projections, int tableNums, int hashSize, // scale of projection planes
    float* bits, // bits for hash
    float* hash_values){

    /*
    initialize gpu memory
    */

    // Initialize the inputs on device
    int input_bytes = inputSize * inputDim * sizeof(float);
    float* gpu_input;
    hipMalloc(&gpu_input, input_bytes);
    hipMemcpy(gpu_input, input, input_bytes, hipMemcpyHostToDevice);

    // Initialize projection planes on device
    int projection_plane_bytes = tableNums * inputDim * hashSize * sizeof(float);
    float* gpu_projections;
    hipMalloc(&gpu_projections, projection_plane_bytes);
    hipMemcpy(gpu_projections, projections, projection_plane_bytes, hipMemcpyHostToDevice);

    // Initialize bits on device
    float* gpu_bits;
    int bits_bytes = hashSize * sizeof(float);
    hipMalloc(&gpu_bits, bits_bytes);
    hipMemcpy(gpu_bits, bits, bits_bytes, hipMemcpyHostToDevice);

    // initialize result
    int results_bytes = tableNums * inputSize * hashSize * sizeof(float);
    float* results = (float*) malloc(results_bytes);
    float* gpu_results;
    hipMalloc(&gpu_results, results_bytes);

    // initialize hash values
    int hash_values_bytes = tableNums * inputSize * sizeof(float);
    float* gpu_hash_values;
    hipMalloc(&gpu_hash_values, hash_values_bytes);
    hipMemcpy(gpu_hash_values, hash_values, hash_values_bytes, hipMemcpyHostToDevice);

    /*
    Start indexing
    */

    dim3 blockSize(2, 512);
    dim3 gridSize((hashSize * tableNums + blockSize.x - 1) / blockSize.x, (inputSize + blockSize.y - 1) / blockSize.y);
    hash_<<<gridSize, blockSize>>>(gpu_input, inputSize, inputDim, gpu_projections, inputDim, hashSize, gpu_results, inputSize, hashSize * tableNums, gpu_bits);

    hipDeviceSynchronize();
    hipError_t cudaStatus = hipGetLastError();
    cout << "Hashing : " << hipGetErrorString(cudaStatus) << endl;

    dim3 blockSizeSum(1, 1024);
    dim3 gridSizeSum(tableNums, (inputSize + blockSizeSum.y - 1) / blockSizeSum.y);
    vec_sum<<<gridSizeSum, blockSizeSum>>>(gpu_results, gpu_hash_values, inputSize, tableNums, hashSize);
    
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    cout << "Sum bits: " << hipGetErrorString(cudaStatus) << endl;

    hipMemcpy(hash_values, gpu_hash_values, hash_values_bytes, hipMemcpyDeviceToHost);

    free(results);

    hipFree(gpu_input);
    hipFree(gpu_projections);
    hipFree(gpu_bits);
    hipFree(gpu_results);
    hipFree(gpu_hash_values);
}

// void indexing(float* input, int inputSize, int inputDim, // scale of input
//     float* projections, int tableNums, int hashSize, // scale of projection planes
//     float* bits, // bits for hash
//     float* hash_values) {
//     index(input, inputSize, inputDim, projections, tableNums, hashSize, bits, hash_values);
// }



